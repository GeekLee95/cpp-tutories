
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add (int a, int b, int *c) {
    *c = a + b;
    printf("into add");
}
int main(int argv, char** argc ) {
    printf( "Hello, World!\n" );
    int c;
    int *device_c;
    hipMalloc((void**)&device_c, sizeof(int));
    add<<<1, 1>>>(2, 7, device_c);
    hipMemcpy(&c, device_c, sizeof(int),
               hipMemcpyDeviceToHost);
    printf("2 + 7 = %d\n", c);
    hipFree(device_c); 
    return 0;
}
